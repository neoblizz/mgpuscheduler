#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <iostream>
#include <thread>
#include <fstream>
#include <algorithm>

#include "matrixmultiply.cuh"
#include "scheduler.cuh"

/**
* @brief Macro for error checking for all GPU calls
* @param[in] ans	The GPU call itself, which evaluates to the hipError_t returned.
*/
#ifndef ERROR_CHECK
#define ERROR_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file,
  int line, bool abort = true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "Cuda error in file '%s' in line '%d': %s\n",
      file, line, hipGetErrorString(code));
    if (abort) exit(code);
  }
}
#endif

__global__ void GPUMatrixMultiply(const int matSize, float * A, float * B, float * C)
{
    // NOTE: shared[] is used to hold both sh_A and sh_B.
    extern __shared__ float shared[];
    unsigned int blockWidth = blockDim.x;
    unsigned int sharedOffsetB = blockWidth*blockWidth;

    float *sh_A = &shared[0];
    float *sh_B = &shared[sharedOffsetB];
    
    unsigned int rowC = blockWidth * blockIdx.y + threadIdx.y;
    unsigned int colC = blockWidth * blockIdx.x + threadIdx.x;

    float temp = 0;
    unsigned int sharedRow = threadIdx.y;
    unsigned int sharedCol = threadIdx.x;
    unsigned int posShared = sharedRow * blockWidth + sharedCol;
    unsigned int posA, posB; // When they're separate, below

    #pragma unroll
    //for (int m = 0; m < (matSize-1)/blockWidth+1; m++)
    for (int block = 0; block < gridDim.x; ++block)
    {
      unsigned int blockOffset = block*blockWidth; // For A this is the column offset, for B, the row offset
      if (rowC < matSize && blockOffset + threadIdx.x < matSize)
        sh_A[posShared] = A[rowC*matSize + (blockOffset + threadIdx.x)];
      else sh_A[posShared] = 0.0f; // Not sure about this
      if (colC < matSize && blockOffset + threadIdx.y < matSize)
        sh_B[posShared] = B[(blockOffset + threadIdx.y) * matSize + colC];
      else sh_B[posShared] = 0.0f; // Not sure about this

     __syncthreads();

      for (int k = 0; k < blockWidth; k++) {
        posA = sharedRow * blockWidth + k;
        posB = k * blockWidth + sharedCol;
        temp += sh_A[posA] * sh_B[posB];
      }
     __syncthreads();
    }

    if (rowC < matSize && colC < matSize)
      C[rowC*matSize + colC] = temp;
}


void MatrixMultiply::FreeHostMemory()
{
  /* 2D Memory release
  for (int i = 0; i < m_vectorSize; i++) {
    if (m_hA[i]) free(m_hA[i]);
    if (m_hB[i]) free(m_hB[i]);
    if (m_hC[i]) free(m_hC[i]);
    if (m_hCheckC[i]) free(m_hCheckC[i]);
  }*/

  if (m_hA) free(m_hA);
  if (m_hB) free(m_hB);
  if (m_hC) free(m_hC);
  if (m_hCheckC) free(m_hCheckC);
  m_hA = m_hB = m_hC = m_hCheckC = NULL;
}

void MatrixMultiply::FreeDeviceMemory()
{
  if (m_dA) ERROR_CHECK(hipFree(m_dA));
  if (m_dB) ERROR_CHECK(hipFree(m_dB));
  if (m_dC) ERROR_CHECK(hipFree(m_dC));
  m_dA = m_dB = m_dC = NULL;
}

/**
* @brief Initialize host vectors for a single MatrixMultiply run.
* @param[in] vectorSize	The size of each vector.
*/
void MatrixMultiply::InitializeData(int matrixSize, int blockWidth, int kernelNum)
{
  m_matrixSize = matrixSize;
  m_blockWidth = blockWidth;
  m_kernelNum = kernelNum;

  m_hA = (float*) malloc(sizeof(float) * matrixSize * matrixSize);
  m_hB = (float*) malloc(sizeof(float) * matrixSize * matrixSize);
  m_hC = (float*) malloc(sizeof(float) * matrixSize * matrixSize);
  m_hCheckC = (float*) malloc(sizeof(float) * matrixSize * matrixSize);

  m_blocksRequired = matrixSize % blockWidth == 0 ? (matrixSize / blockWidth) : 1 + (matrixSize / blockWidth); 
  m_globalMemRequired = 3 * sizeof(float) * matrixSize * matrixSize;

  // Floating pt ops (M=multiply, A=add):
  // Per thread (matrixSize * matrixSize):
  //  2M + 2A for row/col
  //  1M + 1A for posShared
  //  1M + 1A for C index -> TOT = 8
  //  Per block (gridDim.x or m_blocksRequired)
  //    1M for blockOffset
  //    2 * 1A for row check / col check
  //    2 * (1M + 2A) for A/B index -> TOT = 9
  //    Per block thread (blockWidth)
  //      1M + 1A for posA
  //      1M + 1A for posB
  //      1M + 1A for temp -> TOT = 6
  
  m_floatingPointOps = (float)(matrixSize * matrixSize * (8 + m_blocksRequired * (9 + blockWidth * 6)));
  
  // Float memory accesses (R=read, W=write)
  // Per thread (matrixSize * matrixSize):
  //  1W -> TOT = 1
  //  Per block (gridDim.x or m_blocksRequired)
  //    1R + 1W for A to sh_A
  //    1R + 1W for B to sh_B -> TOT = 4
  //    Per block thread (blockWidth)
  //      2R for temp -> TOT = 2

  // Remember to first multiply by 4 for floats
  m_memBytesReadWrite = 4.0f * (matrixSize * matrixSize * (1 + m_blocksRequired * (4 + blockWidth * 2)));

  ERROR_CHECK(hipStreamCreate(&m_stream));
  ERROR_CHECK(hipEventCreate(&m_startQueueEvent));
  ERROR_CHECK(hipEventCreate(&m_startExecEvent));
  ERROR_CHECK(hipEventCreate(&m_finishExecEvent));
  ERROR_CHECK(hipEventCreate(&m_startCudaMallocEvent));
  ERROR_CHECK(hipEventCreate(&m_finishDownloadEvent));

  //float invRandMax = 1000.0f / RAND_MAX; // Produces random numbers between 0 and 1000
  for (int i = 0; i < matrixSize*matrixSize; i++) {
    m_hA[i] = 2.0f; // std::rand() * invRandMax;
    m_hB[i] = 1.0f; // std::rand() * invRandMax;
    //m_hA[i] = std::rand() * invRandMax; // This doesn't work...
    //m_hB[i] = std::rand() * invRandMax;
    m_hCheckC[i] = 0.0f;
  }

  for (int x = 0; x < matrixSize; x++) { // row number of output
    for (int y = 0; y < matrixSize; y++) { // column number of output
        for (int z = 0; z < matrixSize; z++) { // four elements are added for this output
            m_hCheckC[matrixSize*x+y] += m_hA[matrixSize*x+z] * m_hB[matrixSize*z+y];
        }
    }
}

}

/**
* @brief Find a device with enough resources, and if available, decrement the available resources and return the id.
*/
int MatrixMultiply::AcquireDeviceResources(std::vector< DeviceInfo > *deviceInfo)
{
  // Lock this method
  std::lock_guard< std::mutex > guard(m_deviceInfoMutex); // Automatically unlocks when destroyed

  int deviceNum, freeDeviceNum = -1;
  for (deviceNum = 0; deviceNum < (int)deviceInfo->size(); ++deviceNum)
  {
    DeviceInfo &device = deviceInfo->operator[](deviceNum);
    if (m_globalMemRequired < device.m_remainingGlobalMem && m_blocksRequired < device.m_remainingBlocksDimX)
    {
      freeDeviceNum = deviceNum;
      device.m_remainingGlobalMem -= m_globalMemRequired;
      device.m_remainingBlocksDimX -= m_blocksRequired;
      break;
    }
  }

  return freeDeviceNum;
}

/**
* @brief Execution is complete, release the GPU resources for other threads.
*/
void MatrixMultiply::ReleaseDeviceResources(std::vector< DeviceInfo > *deviceInfo)
{
  // Lock this method
  std::lock_guard< std::mutex > guard(m_deviceInfoMutex); // Automatically unlocks when destroyed

  if (Scheduler::m_verbose) std::cout << "** Kernel " << m_kernelNum << " released GPU " << m_deviceNum << " **\n";

  DeviceInfo &device = deviceInfo->operator[](m_deviceNum);
  device.m_remainingGlobalMem += m_globalMemRequired;
  device.m_remainingBlocksDimX += m_blocksRequired;

  // Result is already in host memory, so free GPU memory
  FreeDeviceMemory();
}

/**
* @brief Execution is complete. Record completion event and timers, verify result, and free host memory.
*/
void MatrixMultiply::FinishHostExecution()
{
  // Update timers
  ERROR_CHECK(hipEventElapsedTime(&m_queueTimeMS, m_startQueueEvent, m_startCudaMallocEvent));
  ERROR_CHECK(hipEventElapsedTime(&m_kernelExecTimeMS, m_startExecEvent, m_finishExecEvent));
  ERROR_CHECK(hipEventElapsedTime(&m_totalExecTimeMS, m_startCudaMallocEvent, m_finishDownloadEvent));

  // Compute MFLOP/s, MB/s for this kernel
  m_MFLOPs = m_floatingPointOps / ((2 ^ 20) * (1000 * m_kernelExecTimeMS));
  m_MBps = m_memBytesReadWrite / ((2 ^ 20) * (1000 * m_kernelExecTimeMS));

  // Verify the result
  bool correct(true);
  for (int m = 0; m < m_matrixSize*m_matrixSize; m++) {
    correct = correct && (ceil(m_hC[m]) == ceil(m_hCheckC[m]));
  }

  if (Scheduler::m_verbose) printf("Kernel %d >> Device: %d, Queue: %.3fms, Kernel: %.3fms, Total: %.3fms, MFLOP/s: %.2f, MB/s: %.2f, Correct: %s\n",
    m_kernelNum, m_deviceNum, m_queueTimeMS, m_kernelExecTimeMS, m_totalExecTimeMS, m_MFLOPs, m_MBps, correct ? "True" : "False");

  // Free memory
  FreeHostMemory();
}

/**
* @brief Generate data for the entire batch of MatrixMultiply's being run.
*/
void BatchMatrixMultiply::GenerateData()
{
  m_data.resize(m_batchSize);

  // Get a random generator with a normal distribution, mean = meanVectorSize, stdDev = 0.1*meanVectorSize
  std::normal_distribution< float > normalDist((float)m_meanMatrixSize, 0.1f*m_meanMatrixSize);

  // Seed by the batch size for both the std::rand generator and the std::default_random_engine, used by distribution
  std::srand(m_batchSize);
  std::default_random_engine randomGen(m_batchSize);

  if (Scheduler::m_verbose) std::cout << "** Generating data **\n\tBatch Size: " << m_batchSize << ", Matrix Size: "
    << m_meanMatrixSize << ", Block Width: " << m_blockWidth << "\n";

  for (int kernelNum = 0; kernelNum < m_batchSize; ++kernelNum)
  {
    m_data[kernelNum] = new MatrixMultiply;
    m_data[kernelNum]->InitializeData((int)normalDist(randomGen), m_blockWidth, kernelNum);
  }

  if (Scheduler::m_verbose) std::cout << "** Done generating data **\n\n";
}

void BatchMatrixMultiply::ComputeBatchResults()
{ 
  // Sum up the per-kernel floating point ops and mem bytes accessed
  m_batchFloatingPointOps = m_batchMemBytesReadWrite = 0;
  for (int kernel = 0; kernel < (int)m_data.size(); ++kernel)
  {
    m_batchFloatingPointOps += m_data[kernel]->m_floatingPointOps;
    m_batchMemBytesReadWrite += m_data[kernel]->m_memBytesReadWrite;
  }

  // Use queue times to find which kernel was run first, and which last.
  struct MatrixMultiplyComp
  {
    bool operator()(const MatrixMultiply *lhs, const MatrixMultiply *rhs)
    {
      return lhs->m_queueTimeMS < rhs->m_queueTimeMS;
    }
  };

  std::sort(m_data.begin(), m_data.end(), MatrixMultiplyComp());

  m_batchKernelExecTimeMS = m_batchTotalExecTimeMS = -1;
  m_batchGFLOPs = m_batchGBps = -1;
  if (m_data.size() < 2)
    return;

  const MatrixMultiply &firstKernel = **m_data.begin();
  const MatrixMultiply &lastKernel = **m_data.rbegin();
  ERROR_CHECK(hipEventElapsedTime(&m_batchKernelExecTimeMS, firstKernel.m_startExecEvent, lastKernel.m_finishExecEvent));
  ERROR_CHECK(hipEventElapsedTime(&m_batchTotalExecTimeMS, firstKernel.m_startCudaMallocEvent, lastKernel.m_finishDownloadEvent));

  // Compute GFLOP/s, GB/s for this batch
  m_batchGFLOPs = m_batchFloatingPointOps / ((2 ^ 30) * (1000 * m_batchKernelExecTimeMS));
  m_batchGBps = m_batchMemBytesReadWrite / ((2 ^ 30) * (1000 * m_batchKernelExecTimeMS));
}

void BatchMatrixMultiply::OutputResultsCSV(const std::string &kernelName)
{
  // First output data for each kernel
  std::string filenameKernel = kernelName + std::string("KernelResults.csv");

  // Append in case running from a script (without, file is overwritten)
  std::ofstream csvKernelFile;
  csvKernelFile.open(filenameKernel.c_str(), std::ios::app);

  // Only output header if file is empty
  csvKernelFile.seekp(0, std::ios_base::beg);
  std::size_t posFirst = csvKernelFile.tellp();
  csvKernelFile.seekp(0, std::ios_base::end);
  std::size_t posLast = csvKernelFile.tellp();
  if (posLast-posFirst == 0)
  {
    csvKernelFile << "BatchSize, KernelName, MeanMatrixSize, BlockWidth, MaxDevices, KernelNum, QueueTimeMS"
                  << ", KernelExecTimeMS, TotalExecTimeMS, FloatingPtOps, MemBytes, MFLOPs, MBps\n";
  }

  for (int kernelNum = 0; kernelNum < (int)m_data.size(); ++kernelNum)
  {
    const MatrixMultiply &kernel = *m_data[kernelNum];
    csvKernelFile << m_batchSize << ", " << kernelName.c_str() << ", " << m_meanMatrixSize << ", " << m_blockWidth
                  << ", " << Scheduler::m_maxDevices << ", " << kernel.m_kernelNum << ", " << kernel.m_queueTimeMS
                  << ", " << kernel.m_kernelExecTimeMS << ", " << kernel.m_totalExecTimeMS 
                  << ", " << kernel.m_floatingPointOps << ", " << kernel.m_memBytesReadWrite 
                  << ", " << kernel.m_MFLOPs << ", " << kernel.m_MBps << "\n";
  }

  // Second output data summary for this batch run
  std::string filenameBatch = kernelName + std::string("BatchResults.csv");

  // Append in case running from a script (without, file is overwritten)
  std::ofstream csvBatchFile;
  csvBatchFile.open(filenameBatch.c_str(), std::ios::app);

  // Only output header if file is empty
  csvBatchFile.seekp(0, std::ios_base::beg);
  posFirst = csvBatchFile.tellp();
  csvBatchFile.seekp(0, std::ios_base::end);
  posLast = csvBatchFile.tellp();
  if (posLast - posFirst == 0)
  {
    csvBatchFile << "BatchSize, KernelName, MeanMatrixSize, BlockWidth, MaxDevices, BatchKernelExecTimeMS"
                 << ", BatchTotalExecTimeMS, FloatingPtOps, MemBytes, GFLOPs, GBps\n";
  }

  csvBatchFile << m_batchSize << ", " << kernelName.c_str() << ", " << m_meanMatrixSize << ", " << m_blockWidth
               << ", " << Scheduler::m_maxDevices << ", " << m_batchKernelExecTimeMS 
               << ", " << m_batchTotalExecTimeMS << ", " << m_batchFloatingPointOps 
               << ", " << m_batchMemBytesReadWrite << ", " << m_batchGFLOPs << ", " << m_batchGBps << "\n";
}

// NVCC having trouble parsing the std::thread() call when this is a member function, so keeping it non-member friend
void RunKernelThreaded(BatchMatrixMultiply *batch, int kernelNum)
{
  MatrixMultiply &kernel = *(batch->m_data[kernelNum]);

  // Acquire a GPU
  int deviceNum = -1;
  bool firstAttempt = true;
  while (deviceNum < 0)
  {
    if (firstAttempt)
    {
      if (Scheduler::m_verbose) std::cout << "** Kernel " << kernelNum << " queued for next available GPU **\n";
      firstAttempt = false;
    }

    // Try to acquire GPU resources (using a lock)
    deviceNum = kernel.AcquireDeviceResources(&Scheduler::m_deviceInfo);
  }

  if (Scheduler::m_verbose) std::cout << "** Kernel " << kernelNum << " acquired GPU " << deviceNum << " **\n";

  // Store the device number for use in ReleaseDeviceResources() - not strictly necessary, could be passed in
  kernel.m_deviceNum = deviceNum;

  // Mark the start total execution event
  ERROR_CHECK(hipEventRecord(kernel.m_startCudaMallocEvent, kernel.m_stream));

  // We've got a GPU, use it
  // Allocate memory on the GPU for input and output data
  std::size_t vectorBytes(kernel.m_matrixSize * kernel.m_matrixSize * sizeof(float));
  ERROR_CHECK(hipSetDevice(deviceNum));
  ERROR_CHECK(hipMalloc((void**)&kernel.m_dA, vectorBytes));
  ERROR_CHECK(hipMalloc((void**)&kernel.m_dB, vectorBytes));
  ERROR_CHECK(hipMalloc((void**)&kernel.m_dC, vectorBytes));

  // Upload the input data for this stream
  ERROR_CHECK(hipMemcpyAsync(kernel.m_dA, kernel.m_hA, vectorBytes,
    hipMemcpyHostToDevice, kernel.m_stream));
  ERROR_CHECK(hipMemcpyAsync(kernel.m_dB, kernel.m_hB, vectorBytes,
    hipMemcpyHostToDevice, kernel.m_stream));

  // Mark the start kernel execution event
  ERROR_CHECK(hipEventRecord(kernel.m_startExecEvent, kernel.m_stream));

  // Run the kernel
  size_t sharedMemBytes = 2 * sizeof(float) * kernel.m_blockWidth * kernel.m_blockWidth;
  dim3 dimBlock(kernel.m_blockWidth, kernel.m_blockWidth, 1); // Same dims as other kernel
  dim3 dimGrid(kernel.m_blocksRequired, kernel.m_blocksRequired, 1);
  GPUMatrixMultiply<<<dimGrid, dimBlock, sharedMemBytes, kernel.m_stream >>>(kernel.m_matrixSize, kernel.m_dA, kernel.m_dB, kernel.m_dC);
  ERROR_CHECK(hipPeekAtLastError());

  // Record the time (since stream is non-zero, waits for stream to be complete)
  ERROR_CHECK(hipEventRecord(kernel.m_finishExecEvent, kernel.m_stream));


  // Download the output data for this stream
  ERROR_CHECK(hipMemcpyAsync(kernel.m_hC, kernel.m_dC, vectorBytes,
    hipMemcpyDeviceToHost, kernel.m_stream));

  // Mark the end of total execution event
  ERROR_CHECK(hipEventRecord(kernel.m_finishDownloadEvent, kernel.m_stream));

  // Need to synchronize before releasing resources
  ERROR_CHECK(hipStreamSynchronize(kernel.m_stream));

  // Release the resources (using a lock)
  kernel.ReleaseDeviceResources(&Scheduler::m_deviceInfo);

  // Exiting the function terminates this thread
}

/**
* @brief Run the experiment on a large batch of MatrixMultiply kernels, by using separate CUDA streams per run.
*/
void BatchMatrixMultiply::RunExperiment(const std::string &kernelName)
{
  Scheduler::GetDeviceInfo();
  GenerateData();

  // Mark start queue events (needs to be done here, b/c CPU threads will block eachother)
  for (int kernelNum = 0; kernelNum < (int)m_data.size(); ++kernelNum)
    ERROR_CHECK(hipEventRecord(m_data[kernelNum]->m_startQueueEvent, m_data[kernelNum]->m_stream));

  // Call each kernel instance with a std::thread object
  std::thread *threads = new std::thread[m_data.size()];
  for (int kernelNum = 0; kernelNum < (int)m_data.size(); ++kernelNum)
    threads[kernelNum] = std::thread(RunKernelThreaded, this, kernelNum);

  // Wait for all threads to finish
  for (int kernelNum = 0; kernelNum < (int)m_data.size(); ++kernelNum)
    threads[kernelNum].join();

  // Validate and print results
  if (Scheduler::m_verbose) std::cout << "\n** Kernel Results **\n";
  for (int kernelNum = 0; kernelNum < (int)m_data.size(); ++kernelNum)
  {
    m_data[kernelNum]->FinishHostExecution();
  }

  // Compute accumulated batch results
  ComputeBatchResults();

  // Record results to CSV
  OutputResultsCSV(kernelName);

  ERROR_CHECK(hipDeviceSynchronize());
}
